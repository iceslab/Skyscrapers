#include "hip/hip_runtime.h"
#include "CUDAUtilities.cuh"

size_t desiredFifoSize = CUDA_DEFAULT_FIFO_SIZE;
static bool hasBeenInitialized = false;
extern size_t gpuAlgorithmsToRun;

namespace cuda
{
    hipError_t initDevice(size_t fifoSize)
    {
        hipError_t cudaStatus = hipSuccess;
        if (!hasBeenInitialized)
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?\n");
            }

            if (fifoSize != CUDA_DEFAULT_FIFO_SIZE)
            {
                size_t fifoSizeRef = 0;
                hipDeviceGetLimit(&fifoSizeRef, hipLimitPrintfFifoSize);
                auto converted = bytesToHumanReadable(fifoSizeRef);
                fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());
                converted = bytesToHumanReadable(fifoSize);
                fprintf(stderr, "Setting FIFO size to %5.1f %s\n", converted.first, converted.second.c_str());
                hipDeviceSetLimit(hipLimitPrintfFifoSize, fifoSize);
                hipDeviceGetLimit(&fifoSizeRef, hipLimitPrintfFifoSize);
                converted = bytesToHumanReadable(fifoSizeRef);
                fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());
            }

            hasBeenInitialized = true;
        }
        return cudaStatus;
    }

    hipError_t deinitDevice()
    {
        hipError_t cudaStatus = hipSuccess;
        if (hasBeenInitialized)
        {
            if (--gpuAlgorithmsToRun == 0)
            {
                cudaStatus = hipDeviceReset();
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "hipDeviceReset failed!");
                }
                hasBeenInitialized = false;
            }
        }
        return cudaStatus;
    }

    std::pair<double, std::string> bytesToHumanReadable(double bytes)
    {
        static const std::vector<std::string> postfixes = { "B", "kB", "MB", "GB", "TB", "PB", "EB" };
        const double factor = 1024.0;

        size_t i = 0;
        for (; i < postfixes.size(); i++)
        {
            if (bytes < factor)
            {
                break;
            }
            bytes /= factor;
        }

        return std::make_pair(bytes, postfixes[i]);
    }

    std::pair<double, std::string> timeToHumanReadable(double time, Resolution resolution)
    {
        static const std::vector<std::string> postfixes = { "ns", "us", "ms", "s", "min", "h" };
        static const std::vector<double> factors = { 1000.0, 1000.0, 1000.0, 1000.0, 60.0, 60.0 };

        size_t i = 0;
        switch (resolution)
        {
        case NANOSECONDS:
            i = 0;
            break;
        case MICROSECONDS:
            i = 1;
            break;
        case MILLISECONDS:
            i = 2;
            break;
        case SECONDS:
            i = 3;
            break;
        }

        int direction = 0;

        for (; i > 0 && i < postfixes.size() - 1; i += direction)
        {
            if (time < factors[i + 1] && time >= 1.0)
            {
                break;
            }
            else if (time < 1.0)
            {
                time *= factors[i];
                direction = -1;
            }
            else if (time >= factors[i + 1])
            {
                time /= factors[i + 1];
                direction = 1;
            }
        }

        return std::make_pair(time, postfixes[i]);
    }

    double getTime(int64T start, int64T end, Resolution resolution)
    {
        return getTime(end - start, resolution);
    }

    double getTime(int64T diff, Resolution resolution)
    {
        static int hzClockRate = 0;
        double retVal = std::numeric_limits<double>::quiet_NaN();
        if (hzClockRate == 0)
        {
            int device = 0;
            hipError_t err = hipGetDevice(&device);
            if (err != hipSuccess)
            {
                fprintf(stderr, "hipGetDevice failed! Do you have a CUDA-capable GPU installed?\n");
            }
            else
            {
                hipDeviceProp_t properties;
                err = hipGetDeviceProperties(&properties, device);
                if (err != hipSuccess)
                {
                    fprintf(stderr, "hipGetDeviceProperties failed! Cannot get device #%d properties\n", device);
                }
                else
                {
                    // Clock rate returned in kHz
                    hzClockRate = properties.clockRate * 1000;
                }
            }
        }
        else
        {
            retVal = static_cast<double>(diff) * static_cast<double>(resolution) /
                static_cast<double>(hzClockRate);
        }

        return retVal;
    }
}