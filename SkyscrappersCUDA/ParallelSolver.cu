#include "ParallelSolver.cuh"

namespace cuda
{
    namespace solver
    {

        //CUDA_HOST std::vector<board::Board> generateBoards(const board::Board & board, const size_t stopLevel)
        //{
        //    std::vector<board::Board> retVal;
        //    generateBoards(board, stopLevel, retVal);
        //    return retVal;
        //}

        //CUDA_HOST void generateBoards(const board::Board & board,
        //                              size_t stopLevel,
        //                              std::vector<board::Board> & retVal,
        //                              size_t level,
        //                              size_t row,
        //                              size_t column)
        //{
        //    ASSERT(stopLevel > 0 && stopLevel <= board.size());
        //    DEBUG_CALL(std::cout << "level: " << level << " row: " << row << " column: " << column << "\n";);
        //    DEBUG_CALL(board.print());
        //    const auto treeRowSize = board.size();

        //    // Check if it is last cell
        //    const auto cellPair = getNextFreeCell(row, column);
        //    if (level == stopLevel || cellPair == lastCellPair)
        //    {
        //        retVal.emplace_back(board);
        //    }
        //    else
        //    {
        //        for (size_t i = 0; i < treeRowSize; i++)
        //        {
        //            const auto consideredBuilding = static_cast<board::boardFieldT>(i + 1);
        //            if (board.isBuildingPlaceable(row, column, consideredBuilding))
        //            {
        //                board.setCell(row, column, consideredBuilding);
        //                if (board.isBoardPartiallyValid(row, column))
        //                {
        //                    generateBoards(stopLevel, retVal, level + 1, cellPair.first, cellPair.second);
        //                }

        //                board.clearCell(row, column);
        //            }
        //        }
        //    }
        //}

        CUDA_HOST kernelInputT prepareSolvers(const std::vector<board::Board> & boards, size_t & count)
        {
            // Create array on host
            std::vector<solver::SequentialSolver> prepareRetVal;
            prepareRetVal.reserve(boards.size());

            //for (size_t i = 0; i < boards.size(); i++)
            //{
            //    auto & el = boards[i];
            //    prepareRetVal.push_back(std::move(el));
            //}

            for (auto & el : boards)
            {
                prepareRetVal.push_back(std::move(el));
            }

            // Create array on device
            kernelInputT d_retVal = nullptr;
            count = boards.size();
            hipError_t err = hipMalloc(&d_retVal, count * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
                count = 0;
            }
            else
            {
                // Copy host array to device
                err = hipMemcpy(&d_retVal, prepareRetVal.data(), count * sizeof(*d_retVal), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                    count = 0;
                }
            }

            return d_retVal;
        }

        CUDA_HOST kernelOutputT prepareResultArray(size_t solversCount)
        {
            kernelOutputT d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, solversCount * maxResultsPerThread * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }

            return d_retVal;
        }

        CUDA_HOST kernelOutputSizesT prepareResultArraySizes(size_t solversCount)
        {
            kernelOutputSizesT d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, solversCount * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }

            return d_retVal;
        }

        CUDA_HOST stackT prepareStack(size_t boardSize, size_t solversCount)
        {
            stackT d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, boardSize * boardSize * solversCount * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }

            return d_retVal;
        }

        CUDA_HOST void freeSolvers(kernelInputT & d_solvers)
        {
            hipFree(d_solvers);
            d_solvers = nullptr;
        }

        CUDA_HOST void freeResultArray(kernelOutputT & d_outputBoards)
        {
            hipFree(d_outputBoards);
            d_outputBoards = nullptr;
        }

        CUDA_HOST void freeResultArraySizes(kernelOutputSizesT & d_outputBoardsSizes)
        {
            hipFree(d_outputBoardsSizes);
            d_outputBoardsSizes = nullptr;
        }

        CUDA_HOST void freeStack(stackT & d_stack)
        {
            hipFree(d_stack);
            d_stack = nullptr;
        }

        CUDA_HOST bool verifyAllocation(kernelInputT & d_solvers,
                                        kernelOutputT & d_outputBoards,
                                        kernelOutputSizesT & d_outputBoardsSizes,
                                        stackT & d_stack)
        {
            return d_solvers != nullptr &&
                d_outputBoards != nullptr &&
                d_outputBoardsSizes != nullptr &&
                d_stack != nullptr;
        }

        CUDA_GLOBAL void parallelBoardSolving(const kernelInputT & d_solvers,
                                              kernelOutputT & d_outputBoards,
                                              kernelOutputSizesT & d_outputBoardsSizes,
                                              stackT & d_stack)
        {
            // It denotes thread index and array index
            const auto idx = threadIdx.x;
            d_outputBoardsSizes[idx] = d_solvers[idx].solve(d_outputBoards + idx * maxResultsPerThread, d_stack + idx);
        }
    }
}
