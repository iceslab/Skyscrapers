#include "CUDAUtilities.cuh"

size_t desiredFifoSize = CUDA_DEFAULT_FIFO_SIZE;
static bool hasBeenInitialized = false;
extern size_t gpuAlgorithmsToRun;

namespace cuda
{
    hipError_t initDevice(size_t fifoSize)
    {
        hipError_t cudaStatus = hipSuccess;
        if (!hasBeenInitialized)
        {
            // Choose which GPU to run on, change this on a multi-GPU system.
            cudaStatus = hipSetDevice(0);
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
            }

            if (fifoSize != CUDA_DEFAULT_FIFO_SIZE)
            {
                size_t fifoSizeRef = 0;
                hipDeviceGetLimit(&fifoSizeRef, hipLimitPrintfFifoSize);
                auto converted = bytesToHumanReadable(fifoSizeRef);
                fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());
                converted = bytesToHumanReadable(fifoSize);
                fprintf(stderr, "Setting FIFO size to %5.1f %s\n", converted.first, converted.second.c_str());
                hipDeviceSetLimit(hipLimitPrintfFifoSize, fifoSize);
                hipDeviceGetLimit(&fifoSizeRef, hipLimitPrintfFifoSize);
                converted = bytesToHumanReadable(fifoSizeRef);
                fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());
            }

            hasBeenInitialized = true;
        }
        return cudaStatus;
    }

    hipError_t deinitDevice()
    {
        hipError_t cudaStatus = hipSuccess;
        if (hasBeenInitialized)
        {
            if (--gpuAlgorithmsToRun == 0)
            {
                cudaStatus = hipDeviceReset();
                if (cudaStatus != hipSuccess)
                {
                    fprintf(stderr, "hipDeviceReset failed!");
                }
                hasBeenInitialized = false;
            }
        }
        return cudaStatus;
    }

    std::pair<double, std::string> bytesToHumanReadable(double bytes)
    {
        const std::vector<std::string> postfixes = {"B", "KB", "MB", "GB", "TB", "PB", "EB"};
        const double factor = 1024.0;

        size_t i = 0;
        for (; i < postfixes.size(); i++)
        {
            if (bytes < factor)
            {
                break;
            }
            bytes /= factor;
        }

        return std::make_pair(bytes, postfixes[i]);
    }

}