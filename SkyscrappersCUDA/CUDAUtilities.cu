#include "CUDAUtilities.cuh"

namespace cuda
{
    hipError_t initDevice()
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        hipError_t cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        }

        return cudaStatus;
    }

    hipError_t deinitDevice()
    {
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipDeviceReset failed!");
        }

        return cudaStatus;
    }

}