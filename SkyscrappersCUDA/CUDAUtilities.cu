#include "CUDAUtilities.cuh"

namespace cuda
{
    hipError_t initDevice()
    {
        // Choose which GPU to run on, change this on a multi-GPU system.
        hipError_t cudaStatus = hipSetDevice(0);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        }

        size_t fifoSize = 0;
        hipDeviceGetLimit(&fifoSize, hipLimitPrintfFifoSize);
        auto converted = bytesToHumanReadable(fifoSize);
        fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());
        fifoSize = (512 << 20);
        converted = bytesToHumanReadable(fifoSize);
        fprintf(stderr, "Setting FIFO size to %5.1f %s\n", converted.first, converted.second.c_str());
        hipDeviceSetLimit(hipLimitPrintfFifoSize, fifoSize);
        hipDeviceGetLimit(&fifoSize, hipLimitPrintfFifoSize);
        converted = bytesToHumanReadable(fifoSize);
        fprintf(stderr, "FIFO size (printf): %5.1f %s\n", converted.first, converted.second.c_str());

        return cudaStatus;
    }

    hipError_t deinitDevice()
    {
        hipError_t cudaStatus = hipDeviceReset();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipDeviceReset failed!");
        }

        return cudaStatus;
    }

    std::pair<double, std::string> bytesToHumanReadable(double bytes)
    {
        const std::vector<std::string> postfixes = {"B", "KB", "MB", "GB", "TB", "PB", "EB"};
        const double factor = 1024.0;

        size_t i = 0;
        for (; i < postfixes.size(); i++)
        {
            if (bytes < factor)
            {
                break;
            }
            bytes /= factor;
        }

        return std::make_pair(bytes, postfixes[i]);
    }

}