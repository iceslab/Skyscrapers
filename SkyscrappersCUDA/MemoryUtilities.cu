#include "MemoryUtilities.cuh"

namespace cuda
{
    namespace solver
    {
        CUDA_HOST cuda::uint32T * prepareGeneratedSolversCount(cuda::uint32T generatedSolversCount)
        {
            uint32T* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }
            else
            {
                // Copy host variable to device
                err = hipMemcpy(d_retVal, &generatedSolversCount, sizeof(*d_retVal), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                }
            }
            return d_retVal;
        }
        CUDA_HOST kernelInputT prepareSolvers(const std::vector<board::Board> & boards,
                                              std::vector<SequentialSolver> & h_solvers,
                                              size_t & count)
        {
            // Create array on host
            h_solvers.reserve(boards.size());

            for (auto & el : boards)
            {
                h_solvers.push_back(el);
            }

            // Create array on device
            kernelInputT d_retVal = nullptr;
            count = boards.size();
            hipError_t err = hipMalloc(&d_retVal, count * sizeof(SequentialSolver));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
                count = 0;
            }
            else
            {
                // Copy host array to device
                err = hipMemcpy(d_retVal, h_solvers.data(), count * sizeof(SequentialSolver), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                    count = 0;
                }
            }

            return d_retVal;
        }

        CUDA_HOST cuda::uint32T * prepareSolversTaken()
        {
            uint32T* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }
            else
            {
                uint32T h_valueToSet = 0;
                // Copy host variable to device
                err = hipMemcpy(d_retVal, &h_valueToSet, sizeof(*d_retVal), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                }
            }
            return d_retVal;
        }

        CUDA_HOST kernelOutputT prepareResultArray(std::vector<cuda::Board> & h_boards,
                                                   size_t solversCount,
                                                   size_t boardSize)
        {
            // Create array on host
            h_boards = std::vector<cuda::Board>(CUDA_MAX_RESULTS,
                                                cuda::Board(boardSize));

            kernelOutputT d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, CUDA_MAX_RESULTS * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }
            else
            {
                // Zero out allocated memory
                err = hipMemcpy(d_retVal,
                                 h_boards.data(),
                                 CUDA_MAX_RESULTS * sizeof(*d_retVal),
                                 hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                }
            }

            return d_retVal;
        }

        CUDA_HOST uint32T * prepareResultArraySize()
        {
            uint32T* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }
            else
            {
                uint32T h_valueToSet = 0;
                // Copy host variable to device
                err = hipMemcpy(d_retVal, &h_valueToSet, sizeof(*d_retVal), hipMemcpyHostToDevice);
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memcpy", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                }
            }
            return d_retVal;
        }

        CUDA_HOST threadLocalsT * prepareThreadLocals(size_t solversCount)
        {
            threadLocalsT* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, solversCount * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }
            else
            {
                // Copy host array to device
                err = hipMemset(d_retVal, 0, solversCount * sizeof(*d_retVal));
                if (err != hipSuccess)
                {
                    CUDA_PRINT_ERROR("Failed memset", err);
                    hipFree(d_retVal);
                    d_retVal = nullptr;
                }
            }

            return d_retVal;
        }

        CUDA_HOST uint32T * prepareScatterArray(size_t solversCount)
        {
            uint32T* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, solversCount * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }

            return d_retVal;
        }

        CUDA_HOST cudaEventsDeviceT * prepareCudaEventDevice(const std::vector<cudaEventsDeviceT>& h_timers)
        {
            cudaEventsDeviceT* d_retVal = nullptr;
            hipError_t err = hipMalloc(&d_retVal, h_timers.size() * sizeof(*d_retVal));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed allocation", err);
                d_retVal = nullptr;
            }

            return d_retVal;
        }

        CUDA_HOST void * prepareStack(SolversEnableE solverType,
                                      size_t generatedSolversCount,
                                      size_t cellsCount)
        {
            void* retVal = nullptr;
            const auto stackElements = generatedSolversCount * cellsCount;
            switch (solverType)
            {
            case PARALLEL_GPU_AOS:
                retVal = cuda::solver::Stack::allocateAOSStack(stackElements);
                break;
            case PARALLEL_GPU_SOA:
                retVal = cuda::solver::Stack::allocateSOAStack(stackElements);
                break;
            }
            return retVal;
        }

        CUDA_HOST void prepareConstantMemory(const board::Board & board)
        {
            // Make continuous memory from vector
            std::vector<cuda::boardFieldT> h_hints(4 * board.size());
            for (size_t i = 0; i < 4; i++)
            {
                memcpy(h_hints.data() + i * board.size(),
                       board.hints[i].data(),
                       board.size() * sizeof(cuda::boardFieldT));
            }

            hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(constantMemoryPtr),
                                                 h_hints.data(),
                                                 4 * board.size() * sizeof(cuda::boardFieldT));
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed copying to symbol", err);
            }
        }

        CUDA_HOST kernelOutputT prepareHostResultArray()
        {
            kernelOutputT h_retVal = reinterpret_cast<kernelOutputT>(
                calloc(CUDA_MAX_RESULTS, sizeof(*h_retVal)));
            if (h_retVal == nullptr)
            {
                HOST_PRINT_ERROR("Failed calloc");
            }

            return h_retVal;
        }

        CUDA_HOST void freeGeneratedSolversCount(cuda::uint32T * d_generatedSolversCount)
        {
            hipFree(d_generatedSolversCount);
            d_generatedSolversCount = nullptr;
        }

        CUDA_HOST void freeSolvers(kernelInputT & d_solvers)
        {
            hipFree(d_solvers);
            d_solvers = nullptr;
        }

        CUDA_HOST void freeSolversTaken(cuda::uint32T *& d_solversTaken)
        {
            hipFree(d_solversTaken);
            d_solversTaken = nullptr;
        }

        CUDA_HOST void freeResultArray(kernelOutputT & d_outputBoards)
        {
            hipFree(d_outputBoards);
            d_outputBoards = nullptr;
        }

        CUDA_HOST void freeResultArraySize(uint32T* & d_outputBoardsSize)
        {
            hipFree(d_outputBoardsSize);
            d_outputBoardsSize = nullptr;
        }

        CUDA_HOST void freeThreadLocals(threadLocalsT *& d_threadLocals)
        {
            hipFree(d_threadLocals);
            d_threadLocals = nullptr;
        }

        CUDA_HOST void freeScatterArray(uint32T *& d_scatterArray)
        {
            hipFree(d_scatterArray);
            d_scatterArray = nullptr;
        }

        CUDA_HOST void freeCudaEventDevice(cudaEventsDeviceT *& d_timers)
        {
            hipFree(d_timers);
            d_timers = nullptr;
        }

        CUDA_HOST void freeStack(void *& d_stack)
        {
            hipFree(d_stack);
            d_stack = nullptr;
        }

        CUDA_HOST void freeHostResultArray(kernelOutputT & h_outputBoards)
        {
            free(h_outputBoards);
            h_outputBoards = nullptr;
        }

        CUDA_HOST void copyResultsArray(kernelOutputT h_outputBoards,
                                        kernelOutputT d_outputBoards,
                                        size_t solversCount)
        {
            hipError_t err = hipMemcpy(h_outputBoards,
                                         d_outputBoards,
                                         CUDA_MAX_RESULTS * sizeof(*h_outputBoards),
                                         hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed memcpy", err);
            }
        }

        CUDA_HOST void copyResultsArraySize(uint32T * h_outputBoardsSize,
                                            uint32T * d_outputBoardsSize)
        {
            hipError_t err = hipMemcpy(h_outputBoardsSize,
                                         d_outputBoardsSize,
                                         sizeof(uint32T),
                                         hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed memcpy", err);
            }
        }

        CUDA_HOST void copyCudaEventDevice(std::vector<cudaEventsDeviceT>& h_timers, cudaEventsDeviceT *& d_timers)
        {
            hipError_t err = hipMemcpy(h_timers.data(),
                                         d_timers,
                                         h_timers.size() * sizeof(*d_timers),
                                         hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                CUDA_PRINT_ERROR("Failed memcpy", err);
            }
        }

        CUDA_HOST bool verifyAllocation(kernelInputT & d_solvers,
                                        kernelOutputT & d_outputBoards,
                                        uint32T* & d_outputBoardsSize)
        {
            return d_solvers != nullptr &&
                d_outputBoards != nullptr &&
                d_outputBoardsSize != nullptr;
        }

        CUDA_HOST int getSharedMemorySize(SolversEnableE solverType)
        {
            int retVal = 0;
            switch (solverType)
            {
            case PARALLEL_GPU_SHM:
                retVal = 32 << 10; // 32 kB;
                break;
            }
            return retVal;
        }

        //CUDA_GLOBAL void parallelBoardSolving(kernelInputT d_solvers,
        //                                      kernelOutputT d_outputBoards,
        //                                      kernelOutputSizesT d_outputBoardsSizes,
        //                                      stackT d_stack)
        //{
        //    // It denotes thread index and array index
        //    const auto idx = threadIdx.x;
        //    printf("Thread idx: %u\n");
        //    //d_outputBoardsSizes[idx] = d_solvers[idx].solve(d_outputBoards + idx * maxResultsPerThread, d_stack + idx);
        //}
    }
}
