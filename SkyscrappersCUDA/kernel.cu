#include "hip/hip_runtime.h"
#include "asserts.h"
#include "Timer.h"
#include "../Skyscrappers/SequentialSolver.h"
#include "../Skyscrappers/ParallelCpuSolver.h"
#include <stdio.h>
#include "XGetopt.h"
#include "Statistics.h"

#include "KernelFunctions.inl"

Statistics launchSequentialSolver(const board::Board & board);
Statistics launchParallelCpuSolver(const board::Board & board);
Statistics launchBaseParallelGpuSolver(const board::Board & board);
Statistics launchAOSStackParallelGpuSolver(const board::Board & board);
Statistics launchSOAStackParallelGpuSolver(const board::Board & board);

int main(int argc, char** argv)
{
    if (ProcessCommandLine(argc, argv) == FALSE)
    {
        // Exit when commandline processing fails
        printUsage();
        return -1;
    }

    printLaunchParameters();
    // Prepare data on host
    board::Board b(1);
    if (loadFromFile == true)
    {
        if (!b.readFromFile(filePath))
        {
            printf("Could not read file \"%s\"", filePath);
            return -2;
        }
        b.calculateHints();
    }
    else
    {
        b.generate(boardDimension);
    }
    b.saveToFile("lastRun.txt");

    printf("\nExpected result:\n");
    printf("==========================\n");
    b.print();
    printf("==========================\n");
    fflush(stdout);
    fflush(stderr);

    // CPU solvers
    const auto sStats = launchSequentialSolver(b);
    const auto pcStats = launchParallelCpuSolver(b);
    const auto bpgStats = launchBaseParallelGpuSolver(b);
    const auto aosgStats = launchAOSStackParallelGpuSolver(b);
    const auto soagStats = launchSOAStackParallelGpuSolver(b);

    sStats.print();
    pcStats.print();
    bpgStats.print();
    aosgStats.print();
    soagStats.print();

    //system("pause");
    return 0;
}

Statistics launchSequentialSolver(const board::Board & board)
{
    Statistics retVal(sequentialSolver);
    auto cMilliseconds = std::numeric_limits<double>::quiet_NaN();
    if (sequentialSolver == true)
    {
        solver::SequentialSolver c(board);
        Timer time;
        time.start();
        const auto cResult = c.solve();
        cMilliseconds = time.stop(Resolution::MILLISECONDS);
    }
    retVal.emplace_back("SequentialSolver solving time: ", cMilliseconds);
    return retVal;
}

Statistics launchParallelCpuSolver(const board::Board & board)
{
    Statistics retVal(parallelCpuSolver);
    auto pcMilliseconds = std::numeric_limits<double>::quiet_NaN();
    if (parallelCpuSolver == true)
    {
        solver::ParallelCpuSolver pc(board);
        Timer time;
        time.start();
        const auto pcResult = pc.solve(desiredBoards);
        pcMilliseconds = time.stop(Resolution::MILLISECONDS);
    }
    retVal.emplace_back("ParallelCpuSolver solving time: ", pcMilliseconds);
    return retVal;
}

Statistics launchBaseParallelGpuSolver(const board::Board & board)
{
    Statistics retVal(baseParallelGpuSolver);
    auto pgMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto initMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deinitMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto generationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto allocationMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deallocationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto kernelLaunchMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto kernelSyncMilliseconds = std::numeric_limits<double>::quiet_NaN();

    if (baseParallelGpuSolver == true)
    {
        Timer time;
        Timer timeInit;
        Timer timeGeneration;
        Timer timeAllocation;
        time.start();
        // Initialize device
        cuda::initDevice();
        initMilliseconds = time.stop(Resolution::MILLISECONDS);

        solver::ParallelSolver ps(board);
        timeGeneration.start();
        const auto boards = ps.generateNBoards(desiredBoards);
        generationMilliseconds = timeGeneration.stop(Resolution::MILLISECONDS);
        size_t generatedSolversCount = 0;

        timeAllocation.start();
        // Host vector for solvers - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::solver::SequentialSolver> h_solvers;
        // Host vector for boards - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::Board> h_boards;

        // Allocating memory on device
        auto d_solvers = cuda::solver::prepareSolvers(boards, h_solvers, generatedSolversCount);
        auto d_outputBoards = cuda::solver::prepareResultArray(h_boards, generatedSolversCount, boards.front().size());
        auto d_outputBoardsSizes = cuda::solver::prepareResultArraySizes(generatedSolversCount);

        // Allocating memory on host
        auto h_outputBoards = cuda::solver::prepareHostResultArray(generatedSolversCount);
        auto h_outputBoardsSizes = cuda::solver::prepareHostResultArraySizes(generatedSolversCount);
        allocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);


        // If allocation was successfull launch kernel
        if (cuda::solver::verifyAllocation(d_solvers, d_outputBoards, d_outputBoardsSizes))
        {
            dim3 numBlocks(1);
            dim3 threadsPerBlock(generatedSolversCount);

            printf("Launching kernel...\n");
            fflush(stdout);
            fflush(stderr);

            Timer kernelTimer;
            kernelTimer.start();
            parallelSolvingBase << <numBlocks, threadsPerBlock >> >
                (d_solvers,
                 d_outputBoards,
                 d_outputBoardsSizes);

            kernelLaunchMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);

            // Check for any errors launching the kernel
            hipError_t cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
            {
                printf("parallelSolvingBase launch failed: %s\n", hipGetErrorString(cudaStatus));
            }
            else
            {
                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                cudaStatus = hipDeviceSynchronize();
                kernelSyncMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);
                printf("Kernel finished\n");
                fflush(stdout);
                fflush(stderr);
                if (cudaStatus != hipSuccess)
                {
                    printf("hipDeviceSynchronize returned %d \"%s\"\n",
                           cudaStatus,
                           hipGetErrorString(cudaStatus));
                }
                else
                {
                    cuda::solver::copyResultsArray(h_outputBoards,
                                                   d_outputBoards,
                                                   generatedSolversCount);
                    cuda::solver::copyResultsArraySizes(h_outputBoardsSizes,
                                                        d_outputBoardsSizes,
                                                        generatedSolversCount);

                    for (size_t i = 0; i < generatedSolversCount; i++)
                    {
                        const auto boardCount = h_outputBoardsSizes[i];
                        DEBUG_PRINTLN("Result boards in thread %zu: %zu - max: %zu",
                                      i,
                                      boardCount,
                                      CUDA_MAX_RESULTS_PER_THREAD);
                        for (size_t j = 0; j < boardCount && j < CUDA_MAX_RESULTS_PER_THREAD; j++)
                        {
                            board::Board b(h_outputBoards[i * CUDA_MAX_RESULTS_PER_THREAD + j].getHostVector());
                            b.calculateHints();
                            b.print();
                        }
                    }
                }
            }
        }


        timeAllocation.start();
        // Dellocating host memory (in reverse order)
        cuda::solver::freeHostResultArraySizes(h_outputBoardsSizes);
        cuda::solver::freeHostResultArray(h_outputBoards);

        // Dellocating device memory (in reverse order)
        cuda::solver::freeResultArraySizes(d_outputBoardsSizes);
        cuda::solver::freeResultArray(d_outputBoards);
        cuda::solver::freeSolvers(d_solvers);
        deallocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);

        timeInit.start();
        // Deinitialize device
        cuda::deinitDevice();
        deinitMilliseconds = timeInit.stop(Resolution::MILLISECONDS);
        pgMilliseconds = time.stop(Resolution::MILLISECONDS);
    }

    retVal.emplace_back("Base ParallelGpuSolver solving time: ", pgMilliseconds);
    retVal.emplace_back("\nDevice initialize time: ", initMilliseconds);
    retVal.emplace_back("Device deinitialize time: ", deinitMilliseconds);
    retVal.emplace_back("\nBoard generation time: ", generationMilliseconds);
    retVal.emplace_back("\nMemory allocation time: ", allocationMilliseconds);
    retVal.emplace_back("Memory deallocation time: ", deallocationMilliseconds);
    retVal.emplace_back("\nKernel launch time: ", kernelLaunchMilliseconds);
    retVal.emplace_back("Kernel synchronize time: ", kernelSyncMilliseconds);
    retVal.emplace_back("Allocation + synchronize + deallocation time: ",
                        allocationMilliseconds + kernelSyncMilliseconds + deallocationMilliseconds);
    return retVal;
}

Statistics launchAOSStackParallelGpuSolver(const board::Board & board)
{
    Statistics retVal(aosParallelGpuSolver);
    auto pgMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto initMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deinitMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto generationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto allocationMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deallocationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto kernelLaunchMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto kernelSyncMilliseconds = std::numeric_limits<double>::quiet_NaN();

    if (aosParallelGpuSolver == true)
    {
        Timer time;
        Timer timeInit;
        Timer timeGeneration;
        Timer timeAllocation;
        time.start();
        // Initialize device
        cuda::initDevice();
        initMilliseconds = time.stop(Resolution::MILLISECONDS);

        solver::ParallelSolver ps(board);
        timeGeneration.start();
        const auto boards = ps.generateNBoards(desiredBoards);
        generationMilliseconds = timeGeneration.stop(Resolution::MILLISECONDS);
        size_t generatedSolversCount = 0;

        timeAllocation.start();
        // Host vector for solvers - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::solver::SequentialSolver> h_solvers;
        // Host vector for boards - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::Board> h_boards;

        // Allocating memory on device
        auto d_solvers = cuda::solver::prepareSolvers(boards, h_solvers, generatedSolversCount);
        auto d_outputBoards = cuda::solver::prepareResultArray(h_boards, generatedSolversCount, boards.front().size());
        auto d_outputBoardsSizes = cuda::solver::prepareResultArraySizes(generatedSolversCount);

        // Allocating memory on host
        auto h_outputBoards = cuda::solver::prepareHostResultArray(generatedSolversCount);
        auto h_outputBoardsSizes = cuda::solver::prepareHostResultArraySizes(generatedSolversCount);

        const auto stackElements = generatedSolversCount * boards.front().getCellsCount();
        auto d_stack = cuda::solver::Stack::allocateAOSStack(stackElements);
        allocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);

        // If allocation was successfull launch kernel
        if (cuda::solver::verifyAllocation(d_solvers, d_outputBoards, d_outputBoardsSizes))
        {
            dim3 numBlocks(1);
            dim3 threadsPerBlock(generatedSolversCount);

            printf("Launching kernel...\n");
            fflush(stdout);
            fflush(stderr);

            Timer kernelTimer;
            kernelTimer.start();
            parallelSolvingAOSStack << <numBlocks, threadsPerBlock >> >
                (d_solvers,
                 d_outputBoards,
                 d_outputBoardsSizes,
                 d_stack);

            kernelLaunchMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);

            // Check for any errors launching the kernel
            hipError_t cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
            {
                printf("parallelSolvingAOSStack launch failed: %s\n", hipGetErrorString(cudaStatus));
            }
            else
            {
                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                cudaStatus = hipDeviceSynchronize();
                kernelSyncMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);
                printf("Kernel finished\n");
                fflush(stdout);
                fflush(stderr);
                if (cudaStatus != hipSuccess)
                {
                    printf("hipDeviceSynchronize returned %d \"%s\"\n",
                           cudaStatus,
                           hipGetErrorString(cudaStatus));
                }
                else
                {
                    cuda::solver::copyResultsArray(h_outputBoards,
                                                   d_outputBoards,
                                                   generatedSolversCount);
                    cuda::solver::copyResultsArraySizes(h_outputBoardsSizes,
                                                        d_outputBoardsSizes,
                                                        generatedSolversCount);

                    for (size_t i = 0; i < generatedSolversCount; i++)
                    {
                        const auto boardCount = h_outputBoardsSizes[i];
                        DEBUG_PRINTLN("Result boards in thread %zu: %zu - max: %zu",
                                      i,
                                      boardCount,
                                      CUDA_MAX_RESULTS_PER_THREAD);
                        for (size_t j = 0; j < boardCount && j < CUDA_MAX_RESULTS_PER_THREAD; j++)
                        {
                            board::Board b(h_outputBoards[i * CUDA_MAX_RESULTS_PER_THREAD + j].getHostVector());
                            b.calculateHints();
                            b.print();
                        }
                    }
                }
            }
        }


        timeAllocation.start();
        cuda::solver::Stack::deallocateAOSStack(d_stack);

        // Dellocating host memory (in reverse order)
        cuda::solver::freeHostResultArraySizes(h_outputBoardsSizes);
        cuda::solver::freeHostResultArray(h_outputBoards);

        // Dellocating device memory (in reverse order)
        cuda::solver::freeResultArraySizes(d_outputBoardsSizes);
        cuda::solver::freeResultArray(d_outputBoards);
        cuda::solver::freeSolvers(d_solvers);
        deallocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);

        timeInit.start();
        // Deinitialize device
        cuda::deinitDevice();
        deinitMilliseconds = timeInit.stop(Resolution::MILLISECONDS);
        pgMilliseconds = time.stop(Resolution::MILLISECONDS);
    }

    retVal.emplace_back("AOS ParallelGpuSolver solving time: ", pgMilliseconds);
    retVal.emplace_back("\nDevice initialize time: ", initMilliseconds);
    retVal.emplace_back("Device deinitialize time: ", deinitMilliseconds);
    retVal.emplace_back("\nBoard generation time: ", generationMilliseconds);
    retVal.emplace_back("\nMemory allocation time: ", allocationMilliseconds);
    retVal.emplace_back("Memory deallocation time: ", deallocationMilliseconds);
    retVal.emplace_back("\nKernel launch time: ", kernelLaunchMilliseconds);
    retVal.emplace_back("Kernel synchronize time: ", kernelSyncMilliseconds);
    retVal.emplace_back("Allocation + synchronize + deallocation time: ",
                        allocationMilliseconds + kernelSyncMilliseconds + deallocationMilliseconds);
    return retVal;
}

Statistics launchSOAStackParallelGpuSolver(const board::Board & board)
{
    Statistics retVal(soaParallelGpuSolver);
    auto pgMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto initMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deinitMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto generationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto allocationMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto deallocationMilliseconds = std::numeric_limits<double>::quiet_NaN();

    auto kernelLaunchMilliseconds = std::numeric_limits<double>::quiet_NaN();
    auto kernelSyncMilliseconds = std::numeric_limits<double>::quiet_NaN();

    if (soaParallelGpuSolver == true)
    {
        Timer time;
        Timer timeInit;
        Timer timeGeneration;
        Timer timeAllocation;
        time.start();
        // Initialize device
        cuda::initDevice();
        initMilliseconds = time.stop(Resolution::MILLISECONDS);

        solver::ParallelSolver ps(board);
        timeGeneration.start();
        const auto boards = ps.generateNBoards(desiredBoards);
        generationMilliseconds = timeGeneration.stop(Resolution::MILLISECONDS);
        size_t generatedSolversCount = 0;

        timeAllocation.start();
        // Host vector for solvers - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::solver::SequentialSolver> h_solvers;
        // Host vector for boards - needed to properly execute destructors
        // It's lifetime ensures that pointers on device are valid during kernel execution
        std::vector<cuda::Board> h_boards;

        // Allocating memory on device
        auto d_solvers = cuda::solver::prepareSolvers(boards, h_solvers, generatedSolversCount);
        auto d_outputBoards = cuda::solver::prepareResultArray(h_boards, generatedSolversCount, boards.front().size());
        auto d_outputBoardsSizes = cuda::solver::prepareResultArraySizes(generatedSolversCount);

        // Allocating memory on host
        auto h_outputBoards = cuda::solver::prepareHostResultArray(generatedSolversCount);
        auto h_outputBoardsSizes = cuda::solver::prepareHostResultArraySizes(generatedSolversCount);

        const auto stackElements = generatedSolversCount * boards.front().getCellsCount();
        auto d_stack = cuda::solver::Stack::allocateSOAStack(stackElements);
        allocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);

        // If allocation was successfull launch kernel
        if (cuda::solver::verifyAllocation(d_solvers, d_outputBoards, d_outputBoardsSizes))
        {
            dim3 numBlocks(1);
            dim3 threadsPerBlock(generatedSolversCount);

            printf("Launching kernel...\n");
            fflush(stdout);
            fflush(stderr);

            Timer kernelTimer;
            kernelTimer.start();
            parallelSolvingSOAStack << <numBlocks, threadsPerBlock >> >
                (d_solvers,
                 d_outputBoards,
                 d_outputBoardsSizes,
                 d_stack);

            kernelLaunchMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);

            // Check for any errors launching the kernel
            hipError_t cudaStatus = hipGetLastError();
            if (cudaStatus != hipSuccess)
            {
                printf("parallelSolvingSOAStack launch failed: %s\n", hipGetErrorString(cudaStatus));
            }
            else
            {
                // hipDeviceSynchronize waits for the kernel to finish, and returns
                // any errors encountered during the launch.
                cudaStatus = hipDeviceSynchronize();
                kernelSyncMilliseconds = kernelTimer.stop(Resolution::MILLISECONDS);
                printf("Kernel finished\n");
                fflush(stdout);
                fflush(stderr);
                if (cudaStatus != hipSuccess)
                {
                    printf("hipDeviceSynchronize returned %d \"%s\"\n",
                           cudaStatus,
                           hipGetErrorString(cudaStatus));
                }
                else
                {
                    cuda::solver::copyResultsArray(h_outputBoards,
                                                   d_outputBoards,
                                                   generatedSolversCount);
                    cuda::solver::copyResultsArraySizes(h_outputBoardsSizes,
                                                        d_outputBoardsSizes,
                                                        generatedSolversCount);

                    for (size_t i = 0; i < generatedSolversCount; i++)
                    {
                        const auto boardCount = h_outputBoardsSizes[i];
                        DEBUG_PRINTLN("Result boards in thread %zu: %zu - max: %zu",
                                      i,
                                      boardCount,
                                      CUDA_MAX_RESULTS_PER_THREAD);
                        for (size_t j = 0; j < boardCount && j < CUDA_MAX_RESULTS_PER_THREAD; j++)
                        {
                            board::Board b(h_outputBoards[i * CUDA_MAX_RESULTS_PER_THREAD + j].getHostVector());
                            b.calculateHints();
                            b.print();
                        }
                    }
                }
            }
        }

        timeAllocation.start();
        cuda::solver::Stack::deallocateSOAStack(d_stack);

        // Dellocating host memory (in reverse order)
        cuda::solver::freeHostResultArraySizes(h_outputBoardsSizes);
        cuda::solver::freeHostResultArray(h_outputBoards);

        // Dellocating device memory (in reverse order)
        cuda::solver::freeResultArraySizes(d_outputBoardsSizes);
        cuda::solver::freeResultArray(d_outputBoards);
        cuda::solver::freeSolvers(d_solvers);
        deallocationMilliseconds = timeAllocation.stop(Resolution::MILLISECONDS);

        timeInit.start();
        // Deinitialize device
        cuda::deinitDevice();
        deinitMilliseconds = timeInit.stop(Resolution::MILLISECONDS);
        pgMilliseconds = time.stop(Resolution::MILLISECONDS);
    }

    retVal.emplace_back("SOA ParallelGpuSolver solving time: ", pgMilliseconds);
    retVal.emplace_back("\nDevice initialize time: ", initMilliseconds);
    retVal.emplace_back("Device deinitialize time: ", deinitMilliseconds);
    retVal.emplace_back("\nBoard generation time: ", generationMilliseconds);
    retVal.emplace_back("\nMemory allocation time: ", allocationMilliseconds);
    retVal.emplace_back("Memory deallocation time: ", deallocationMilliseconds);
    retVal.emplace_back("\nKernel launch time: ", kernelLaunchMilliseconds);
    retVal.emplace_back("Kernel synchronize time: ", kernelSyncMilliseconds);
    retVal.emplace_back("Allocation + synchronize + deallocation time: ",
                        allocationMilliseconds + kernelSyncMilliseconds + deallocationMilliseconds);
    return retVal;
}
