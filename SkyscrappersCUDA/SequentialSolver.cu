#include "hip/hip_runtime.h"
#include "SequentialSolver.cuh"

namespace cuda
{
    namespace solver
    {
        SequentialSolver::SequentialSolver(const board::Board & board) :
            Solver(board),
            maxVal(std::numeric_limits<size_t>::max()),
            lastCellPair(rowAndColumnPairT(maxVal, maxVal))
        {
            this->board.clear();
        }

        CUDA_DEVICE size_t SequentialSolver::solve(cuda::Board* resultArray, stackT stack)
        {

#ifdef BT_WITH_STACK
            auto retVal = backTrackingWithStack(resultArray, stack);
#else
            auto freeCell = rowAndColumnPairT(0, 0);
            if (board.getCell(0, 0) != 0)
            {
                freeCell = getNextFreeCell(0, 0);
            }

            std::vector<board::Board> retVal;
            backTracking(retVal, 0, freeCell.first, freeCell.second);
#endif // BT_WITH_STACK

            return retVal;
        }

#ifndef BT_WITH_STACK
        void solver::SequentialSolver::backTracking(std::vector<board::Board> & retVal, size_t level, size_t row, size_t column)
        {
            const auto treeRowSize = board.size();

            // Check if it is last cell
            const auto cellPair = getNextFreeCell(row, column);
            if (cellPair == lastCellPair)
            {
                DEBUG_PRINTLN_VERBOSE_INFO("Last cell");
                for (size_t i = 0; i < treeRowSize; i++)
                {
                    const auto consideredBuilding = i + 1;

                    if (board.isBuildingPlaceable(row, column, consideredBuilding))
                    {
                        board.setCell(row, column, consideredBuilding);
                        if (board.isBoardPartiallyValid(row, column))
                        {
                            DEBUG_PRINTLN_VERBOSE_INFO("Found result");
                            DEBUG_CALL(board.print());
                            retVal.emplace_back(board);
                        }
                        board.clearCell(row, column);
                    }
                }
            }
            else
            {
                for (size_t i = 0; i < treeRowSize; i++)
                {
                    const auto consideredBuilding = i + 1;
                    if (board.isBuildingPlaceable(row, column, consideredBuilding))
                    {
                        board.setCell(row, column, consideredBuilding);
                        if (board.isBoardPartiallyValid(row, column))
                        {
                            backTracking(retVal, level + 1, cellPair.first, cellPair.second);
                        }

                        board.clearCell(row, column);
                    }
                }
            }
        }
#else
        CUDA_DEVICE size_t SequentialSolver::backTrackingWithStack(cuda::Board* resultArray, stackT stack)
        {
            size_t resultsCount = 0;
            size_t stackSize = 0;

            rowAndColumnPairT initialCellPair(0, 0);
            if (board.getCell(0, 0) != 0)
            {
                initialCellPair = getNextFreeCell(0, 0);
            }

            auto stackEntrySize = board.getSize();
            stack[stackSize].first.clearAll();
            stack[stackSize++].second = initialCellPair;
            do
            {
                auto & entry = stack[stackSize - 1].first;
                auto & stackCell = stack[stackSize - 1].second;

                const auto & row = stackCell.first;
                const auto & column = stackCell.second;

                auto idx = entry.firstZero();
                if (idx != entry.badIndex)
                {
                    entry.setBit(idx);

                    const auto consideredBuilding = idx + 1;
                    if (board.isBuildingPlaceable(row, column, consideredBuilding))
                    {
                        board.setCell(row, column, consideredBuilding);
                        if (board.isBoardPartiallyValid(row, column))
                        {
                            const auto nextCellPair = getNextFreeCell(row, column);
                            if (nextCellPair == lastCellPair)
                            {
                                if (resultsCount < maxResultsPerThread)
                                {
                                    memcpy(resultArray + resultsCount++, &board, sizeof(board));
                                }
                                else
                                {
                                    // Found a result, but it doesn't fit inside array
                                }
                                board.clearCell(row, column);
                            }
                            else
                            {
                                stack[stackSize].first.clearAll();
                                stack[stackSize++].second = nextCellPair;
                            }
                        }
                        else
                        {
                            board.clearCell(row, column);
                        }
                    }
                }
                else
                {
                    board.clearCell(row, column);
                    --stackSize;
                    if (stackSize > 0)
                    {
                        const auto & newStackCell = stack[stackSize - 1].second;
                        board.clearCell(newStackCell.first, newStackCell.second);
                    }
                }

            } while (stackSize > 0);

            return resultsCount;
        }
#endif // !BT_WITH_STACK

        CUDA_DEVICE rowAndColumnPairT SequentialSolver::getNextFreeCell(size_t row, size_t column) const
        {
            const auto maxSize = board.getSize();

            // Search till free cell is found
            do
            {
                // Next column
                if (column < maxSize - 1)
                {
                    column++;
                }
                // Next row
                else if (column >= maxSize - 1)
                {
                    column = 0;
                    row++;
                }
            } while (row < maxSize && board.getCell(row, column) != 0);

            // If row is too big return max values
            if (row >= maxSize)
            {
                row = maxVal;
                column = maxVal;
            }

            return rowAndColumnPairT(row, column);
        }
    }
}
